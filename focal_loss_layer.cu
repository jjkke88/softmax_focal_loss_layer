#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/focal_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void LogOpGPU(const int nthreads,
          const Dtype* in, Dtype* out, const Dtype eps)
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    out[index] = log(max(in[index], eps));
  }
}

template <typename Dtype>
void SoftmaxFocalLossLayer<Dtype>::compute_intermediate_values_of_gpu() {
  // compute the corresponding variables
  const int count        = prob_.count();
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* ones_data = ones_.gpu_data();
  Dtype* log_prob_data   = log_prob_.mutable_gpu_data();
  Dtype* power_prob_data = power_prob_.mutable_gpu_data();

  /// log(p_t)
  const int nthreads     = prob_.count();
  const Dtype eps        = Dtype(FLT_MIN); // where FLT_MIN = 1.17549e-38, here u can change it
  // more stable
  // NOLINT_NEXT_LINE(whitespace/operators)
  LogOpGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, log_prob_data, eps);
  /// caffe_gpu_log(count,  prob_data, log_prob_data);

  /// (1 - p_t) ^ gamma
  caffe_gpu_sub(count,  ones_data, prob_data, power_prob_data);
  caffe_gpu_powx(count, power_prob_.gpu_data(), gamma_, power_prob_data);
  caffe_gpu_scal(count, alpha_, power_prob_data);
}

template <typename Dtype>
__global__ void FocalLossForwardGPU(const int nthreads,
          const Dtype* log_prob_data, 
          const Dtype* power_prob_data,
          const Dtype* label, 
          Dtype* loss,
          const int num, 
          const int dim, 
          const int spatial_dim,
          const bool has_ignore_label_, 
          const int ignore_label_,
          Dtype* counts) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index]   = 0;
      counts[index] = 0;
    } else {
      int ind       = n * dim + label_value * spatial_dim + s;
      // loss[index]   = -max(power_prob_data[ind] * log_prob_data[ind], Dtype(log(Dtype(FLT_MIN))));
      loss[index]   = -power_prob_data[ind] * log_prob_data[ind];
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxFocalLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
  // The forward pass computes the softmax prob values.
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);

  // compute all needed values
  compute_intermediate_values_of_gpu();

  // const Dtype* prob_data       = prob_.gpu_data();
  const Dtype* log_prob_data   = log_prob_.gpu_data();
  const Dtype* power_prob_data = power_prob_.gpu_data();
  const Dtype* label           = bottom[1]->gpu_data();
  const int dim                = prob_.count() / outer_num_;
  const int nthreads           = outer_num_ * inner_num_;

  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();

  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();

  // NOLINT_NEXT_LINE(whitespace/operators)
  FocalLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, log_prob_data, power_prob_data, 
      label, loss_data,outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);

  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  Dtype valid_count = -1;

  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(nthreads, counts, &valid_count);
  }
  top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_,
                                                        valid_count);
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void FocalLossBackwardGPU(const int nthreads, 
          const Dtype* top,
          const Dtype* label,
          const Dtype* prob_data,
          const Dtype* log_prob_data,
          const Dtype* power_prob_data,
          Dtype* bottom_diff, 
          const int num, 
          const int dim,
          const int spatial_dim, 
          const Dtype gamma,
          const bool has_ignore_label_,
          const int ignore_label_, 
          const Dtype eps,
          Dtype* counts) 
{
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      // the gradient from FL w.r.t p_t, here ignore the `sign`
      int ind_i  = n * dim + label_value * spatial_dim + s; // index of ground-truth label
      Dtype grad = 0 - gamma * (power_prob_data[ind_i] / max(1 - prob_data[ind_i], eps)) 
                             * log_prob_data[ind_i] * prob_data[ind_i]
                     + power_prob_data[ind_i];
      // the gradient w.r.t input data x
      for (int c = 0; c < channels; ++c) {
        int ind_j = n * dim + c * spatial_dim + s;
        if(c == label_value) {
          // if i == j, (here i,j are refered for derivative of softmax)
          bottom_diff[ind_j] = grad * (prob_data[ind_i] - 1);
        } else {
          // if i != j, (here i,j are refered for derivative of softmax)
          bottom_diff[ind_j] = grad * prob_data[ind_j];
        }
      }
      // count
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxFocalLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff     = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data  = top[0]->gpu_data();
    const Dtype* label     = bottom[1]->gpu_data();
    const int dim          = prob_.count() / outer_num_;
    const int nthreads     = outer_num_ * inner_num_;
    const Dtype eps        = 1e-10;

    // intermidiate  
    const Dtype* log_prob_data   = log_prob_.gpu_data();
    const Dtype* power_prob_data = power_prob_.gpu_data();

    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();

    // NOLINT_NEXT_LINE(whitespace/operators)
    FocalLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, prob_data, log_prob_data, power_prob_data,
        bottom_diff, outer_num_, dim, inner_num_, gamma_, has_ignore_label_, ignore_label_, eps, counts);

    // Only launch another CUDA kernel if we actually need the count of valid outputs.
    Dtype valid_count = -1;
    if (normalization_ == LossParameter_NormalizationMode_VALID &&
        has_ignore_label_) {
      caffe_gpu_asum(nthreads, counts, &valid_count);
    }
    // Scale gradient
    const Dtype loss_weight = top[0]->cpu_diff()[0] / get_normalizer(normalization_, valid_count);
    caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxFocalLossLayer);

}  // namespace caffe
